#include <utility>
#include <memory>
#include <string>
#include <atomic>
#include <type_traits>
#include <cassert>

#include <iostream>
#include <iomanip>

#include <hip/hip_runtime.h>

using std::exception;
using std::string;
using std::unique_ptr;
using std::shared_ptr;
using std::atomic;
using std::uint32_t;

struct cuda_rt_exception : exception
{
  cuda_rt_exception(hipError_t error_, char const* message_)
  // {{{
    : error(error_)
    , message(
        string(hipGetErrorName(error_)) + ": "
      + hipGetErrorString(error_) + ": "
      + message_
      )
  {}
  // }}}

  hipError_t code() const
  { // {{{
    return error;
  } // }}}

  virtual const char* what() const noexcept
  { // {{{
    return message.c_str();
  } // }}}

private:
  hipError_t error;
  string message;
};

struct cuda_drv_exception : exception
{
  cuda_drv_exception(hipError_t error_, char const* message_)
  // {{{
    : error(error_)
  {
    const char* str = nullptr;
    hipDrvGetErrorName(error_, &str);
    message = str;
    message += ": ";
    hipDrvGetErrorString(error_, &str);
    message += str;
    message += ": ";
    message += message_;
  }
  // }}}

  hipError_t code() const
  { // {{{
    return error;
  } // }}}

  virtual const char* what() const noexcept
  { // {{{
    return message.c_str();
  } // }}}

private:

  hipError_t error;
  string message;
};

#if defined(__GNUC__) || (defined(__MWERKS__) && (__MWERKS__ >= 0x3000)) || (defined(__ICC) && (__ICC >= 600)) || defined(__ghs__)
  #define CURRENT_FUNCTION __PRETTY_FUNCTION__
#elif defined(__DMC__) && (__DMC__ >= 0x810)
  #define CURRENT_FUNCTION __PRETTY_FUNCTION__
#elif defined(__FUNCSIG__)
  #define CURRENT_FUNCTION __FUNCSIG__
#elif (defined(__INTEL_COMPILER) && (__INTEL_COMPILER >= 600)) || (defined(__IBMCPP__) && (__IBMCPP__ >= 500))
  #define CURRENT_FUNCTION __FUNCTION__
#elif defined(__BORLANDC__) && (__BORLANDC__ >= 0x550)
  #define CURRENT_FUNCTION __FUNC__
#elif defined(__STDC_VERSION__) && (__STDC_VERSION__ >= 199901)
  #define CURRENT_FUNCTION __func__
#elif defined(__cplusplus) && (__cplusplus >= 201103)
  #define CURRENT_FUNCTION __func__
#else
  #define CURRENT_FUNCTION "(unknown)"
#endif

__host__
inline void throw_on_cuda_rt_error(hipError_t error, char const* message)
{ // {{{
  if (hipSuccess != error)
    throw cuda_rt_exception(error, message);
} // }}}

#define THROW_ON_CUDA_RT_ERROR(error) throw_on_cuda_rt_error(error, CURRENT_FUNCTION)

__host__
inline void throw_on_cuda_drv_error(hipError_t error, char const* message)
{ // {{{
  if (hipSuccess != error)
    throw cuda_drv_exception(error, message);
} // }}}

#define THROW_ON_CUDA_DRV_ERROR(error) throw_on_cuda_drv_error(error, CURRENT_FUNCTION)

struct cuda_stream_deleter final
{
  __host__
  inline void operator()(hipStream_t s) const
  { // {{{
    if (nullptr != s)
      THROW_ON_CUDA_RT_ERROR(hipStreamDestroy(s));
  } // }}}
};

struct hip_stream final {
  hip_stream() {
    ihipStream_t* s;
    THROW_ON_CUDA_RT_ERROR(hipStreamCreate(&s));
    ptr.reset(s, cuda_stream_deleter{});
  }

  hip_stream(hip_stream const&) = default;
  hip_stream(hip_stream&&) = default;

  ihipStream_t* operator->() const {
    return ptr.get();
  }

  ihipStream_t* get() const {
    return ptr.get();
  }

private:
  shared_ptr<ihipStream_t> ptr;
};

template <typename T>
struct cuda_free_deleter final
{
  __host__
  inline void operator()(T* p) const
  { // {{{
    if (nullptr != p)
    {
      p->~T();
      THROW_ON_CUDA_RT_ERROR(hipFree(p));
    }
  } // }}}
};

template <typename T>
struct cuda_free_host_deleter final
{
  __host__
  inline void operator()(T* p) const
  { // {{{
    if (nullptr != p)
    {
      p->~T();
      THROW_ON_CUDA_RT_ERROR(hipHostFree((void*)(p)));
    }
  } // }}}
};

template <typename T>
struct cuda_host_pinned_unique_ptr final {
  cuda_host_pinned_unique_ptr() {
    T* r;
    THROW_ON_CUDA_RT_ERROR(hipHostAlloc(&r, sizeof(T), 0));
    new (const_cast<std::remove_cv_t<T>*>(r)) T;
    ptr.reset(r);
  }

  cuda_host_pinned_unique_ptr(T&& t) {
    T* r;
    THROW_ON_CUDA_RT_ERROR(hipHostAlloc(&r, sizeof(T), 0));
    new (const_cast<std::remove_cv_t<T>*>(r)) T(move(t));
    ptr.reset(r);
  }

  cuda_host_pinned_unique_ptr(T const& t) {
    T* r;
    THROW_ON_CUDA_RT_ERROR(hipHostAlloc(&r, sizeof(T), 0));
    new (const_cast<std::remove_cv_t<T>*>(r)) T(t);
    ptr.reset(r);
  }

  cuda_host_pinned_unique_ptr(cuda_host_pinned_unique_ptr const&) = default;
  cuda_host_pinned_unique_ptr(cuda_host_pinned_unique_ptr&&) = default;

  T* operator->() const {
    return ptr.get();
  }

  T* get() const {
    return ptr.get();
  }

private:
  unique_ptr<T, cuda_free_host_deleter<T>> ptr;
};

///////////////////////////////////////////////////////////////////////////////

struct share_stream_t final {};

share_stream_t share_stream{};

struct not_ready_t final {};

not_ready_t not_ready{};

template <typename T>
struct shared_state final
{ // {{{
  int device;
  hip_stream stream;
  cuda_host_pinned_unique_ptr<atomic<uint32_t> volatile> ready;
  unique_ptr<T, cuda_free_deleter<T>> value;

  shared_state()
    : stream()
    , ready(0)
  { // {{{
    THROW_ON_CUDA_RT_ERROR(hipGetDevice(&device));

    T* v;
    THROW_ON_CUDA_RT_ERROR(hipMallocManaged(&v, sizeof(T)));
    new (v) T;
    value.reset(v);
  } // }}}

  shared_state(share_stream_t, shared_state const& other)
    : stream(other.stream)
    , ready(0)
  { // {{{
    THROW_ON_CUDA_RT_ERROR(hipGetDevice(&device));

    T* v;
    THROW_ON_CUDA_RT_ERROR(hipMallocManaged(&v, sizeof(T)));
    new (v) T;
    value.reset(v);
  } // }}}
}; // }}}

template <typename Continuation, typename U, typename T>
__global__
void continuation_kernel(Continuation c, U* u, T* t)
{ // {{{
  *u = c(*t);
} // }}}

struct cuda_executor final
{
  template <typename T>
  struct future;

  template <typename T>
  struct promise final
  { // {{{
    std::shared_ptr<shared_state<T>> ss;

    promise() {}

    promise(future<T> const& f) : ss(f.ss) {}

    void set_value(T&& value) 
    { // {{{
      *ss->value = std::forward<T>(value);
      ss->ready->store(true, std::memory_order_release); 
    } // }}}
  }; // }}}

  template <typename T>
  struct future final
  { // {{{
    using value_type = T;

    std::shared_ptr<shared_state<T>> ss;

    future()
    // {{{
      : ss(std::make_shared<shared_state<T>>())
    {}
    // }}}

    future(not_ready_t)
    // {{{
      : ss(std::make_shared<shared_state<T>>())
    {
      THROW_ON_CUDA_DRV_ERROR(hipStreamWaitValue32(stream(), semaphore(),
                                                  true, hipStreamWaitValueEq));
    }
    // }}}

    future(share_stream_t, future const& other)
    // {{{
      : ss(std::make_shared<shared_state<T>>(share_stream, *other.ss))
    {}
    // }}}

    ihipStream_t* stream() const
    { // {{{
      return ss->stream.get();
    } // }}}

    hipDeviceptr_t semaphore() const
    { // {{{
      return reinterpret_cast<hipDeviceptr_t>(ss->ready.get());
    } // }}}

    T* content() const
    { // {{{
      return ss->value.get();
    } // }}}

    int device() const
    { // {{{
      return ss->device;
    } // }}}

    template <typename Executor>
    auto via(Executor&& exec)
    { // {{{
      auto pf = std::forward<Executor>(exec).make_promise();

      THROW_ON_CUDA_RT_ERROR(hipStreamAddCallback(stream(),
        [=] (ihipStream_t*, hipError_t, void*)
        { pf.first.set_value(*ss->value); }, nullptr, 0));

      return pf.second;
    } // }}}

    template <typename Continuation>
    auto then(Continuation&& c)
    { // {{{
      return cuda_executor{}.then_execute(std::forward<Continuation>(c));
    } // }}}
  }; // }}}

  template <typename T, typename Continuation>
  auto then_execute(future<T> const& prev, Continuation c)
  { // {{{
    using U = decltype(std::declval<Continuation>()(std::declval<T>()));

    int device = 0;
    THROW_ON_CUDA_RT_ERROR(hipGetDevice(&device));

    if (prev.device() == device)
    {
      future<U> next(share_stream, prev);

      continuation_kernel<<<1, 1, 0, next.stream()>>>(c, next.content(), prev.content());
      THROW_ON_CUDA_RT_ERROR(hipGetLastError());

      THROW_ON_CUDA_DRV_ERROR(hipStreamWriteValue32(next.stream(), next.semaphore(),
                                                   true, 0));

      return next;
    }

    else
    {
      future<U> next;

      THROW_ON_CUDA_DRV_ERROR(hipStreamWaitValue32(next.stream(), prev.semaphore(),
                                                  true, hipStreamWaitValueEq));

      continuation_kernel<<<1, 1, 0, next.stream()>>>(c, next.content(), prev.content());
      THROW_ON_CUDA_RT_ERROR(hipGetLastError());

      THROW_ON_CUDA_DRV_ERROR(hipStreamWriteValue32(next.stream(), next.semaphore(),
                                                   true, 0));

      return next;
    }
  } // }}}

  template <typename T>
  std::pair<promise<T>, future<T>> make_promise()
  { // {{{
    future<T>  f(not_ready);
    promise<T> p(f);
    return {std::move(p), std::move(f)};
  } // }}}
};

int main()
{
  cuda_executor exec;

  auto pf = exec.make_promise<int>();
  cuda_executor::promise<int>& p = pf.first;
  cuda_executor::future<int>&  f = pf.second;

  auto g = exec.then_execute(f, [] __host__ __device__ (int x)
                                { printf("%u\n", x); return x + 42; });

  auto h = exec.then_execute(g, [] __host__ __device__ (int x)
                                { printf("%u\n", x); return x + 15; });

  p.set_value(17);

  THROW_ON_CUDA_RT_ERROR(hipDeviceSynchronize());
}

